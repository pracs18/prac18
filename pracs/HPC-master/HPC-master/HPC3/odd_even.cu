#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

__global__ void oddEven(int *d,int I, int n)
{
   int id=threadIdx.x;
   if(I==0 &&((id*2+1)<n))
   {
       if(d[id*2] > d[id*2+1])
       {
             int temp  = d[id*2];
             d[id*2]   = d[id*2+1];
             d[id*2+1] = temp;
   
       }

   }

   if(I==1 &&((id*2+2)<n))
   {
       if(d[id*2+1] > d[id*2+2])
       {
             int temp  = d[id*2+1];
             d[id*2+1]   = d[id*2+2];
             d[id*2+2] = temp;
   
       }

   }



}

int main()
{
   int input[100] , output[100], n, i;
   int *device;
   
   printf("\n\nEnter number of elements :");
   scanf("%d",&n);

   int size=n*sizeof(int);
   hipMalloc(&device,size);
  
   printf("\n\nEnter numbers :");
   for(i=0 ;i<n ;i++)
   {
      scanf("%d",&input[i]);
    
   }
 
   printf("\n\nArray Before Sorting =>");
   for(i=0 ;i<n ;i++)
   {
      printf("%d  ",input[i]);
    
   }

   hipMemcpy(device,input,size,hipMemcpyHostToDevice);


   for(i=0 ;i<n ;i++)
   {
      oddEven<<<1,n>>>(device,i%2,n);
    
   }
   
   hipMemcpy(output,device,size,hipMemcpyDeviceToHost);

   printf("\n\nArray After Sorting =>");
   for(i=0 ;i<n ;i++)
   {
      printf("%d  ",output[i]);
    
   } 

   hipFree(device);
  
   return 0;


}
