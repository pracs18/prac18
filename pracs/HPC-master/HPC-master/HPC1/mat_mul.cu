#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define N 4
#define TPB 2

__global__ void matrixMul(int *a,  int *b,int *c ,int n)
{
   int row = blockIdx.y * blockDim.y + threadIdx.y ;
   int col = blockIdx.x * blockDim.x + threadIdx.x ; 
   int i;
   int sum=0;
   for( i=0 ;i<N; i++)
   {
     sum+= a[row * N+i] * b[i * N+col];
   }
   c[row * N+col] = sum;
}


int main()
{
  int *h_a,*h_b,*h_c,*d_a,*d_b,*d_c;
  
  int size = sizeof(int)*N*N;
  
  h_a = (int*)malloc(size);
  h_b = (int*)malloc(size);
  h_c = (int*)malloc(size);

  hipMalloc(&d_a,size);
  hipMalloc(&d_b,size);
  hipMalloc(&d_c,size);

  int i,j;

  for(i=0; i<N*N;i++)
  {
    
       h_a[i]=random() % N;
       h_b[i]=random() % N; 
    
  } 
  
  printf("\nMatrx A =>\n");
  for(i=0;i<N;i++)
  {
    for(j=0;j<N;j++)
    {
      printf(" %d",h_a[i*N+j]);
    }
    printf("\n");
  }

  printf("\nMatrx B =>\n");
  for(i=0; i<N ;i++)
  {
    for(j=0; j<N ;j++)
    {
      printf(" %d",h_b[i*N+j]);
    }
    printf("\n");
  }

  hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);
   
  int BLOCK_SIZE= N/TPB;
  dim3 GridSize(BLOCK_SIZE,BLOCK_SIZE);
  dim3 BlockSize(TPB, TPB);

  matrixMul<<<GridSize,BlockSize>>>(d_a , d_b ,d_c,N);

  hipMemcpy(h_c,d_c, size, hipMemcpyDeviceToHost);

  printf("\nMatrx C =>\n");
  for(i=0;i<N;i++)
  {
    for(j=0;j<N;j++)
    {
      printf(" %d",h_c[i*N+j]);
    }
    printf("\n");
  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  free(h_a);
  free(h_b);
  free(h_c);

  return 0;

}
