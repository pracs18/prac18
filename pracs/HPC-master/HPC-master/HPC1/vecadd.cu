
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

__global__ void add(double *a,double *b,double *c,int n)
{
  int id=blockIdx.x*blockDim.x+threadIdx.x;

  if(id<n)
  {
   c[id] = a[id] + b[id];
  }


}



int main()
{
  int n=8;
  double *h_a,*h_b,*h_c,*d_a,*d_b,*d_c;
  
  size_t bytes = n*sizeof(double);

  h_a=(double*)malloc(bytes);
  h_b=(double*)malloc(bytes);
  h_c=(double*)malloc(bytes);

  hipMalloc(&d_a,bytes);
  hipMalloc(&d_b,bytes);
  hipMalloc(&d_c,bytes);

  int i;
  for(i=0;i<n;i++)
  {
    h_a[i]= random() %n;
    h_b[i]= random() %n;
  }
  
  printf("\n\nVector A =>");
  for(i=0;i<n;i++)
  {
   
    printf("%lf ",h_a[i]);

  }

  printf("\n\nVector B =>");
  for(i=0;i<n;i++)
  {
   
    printf("%lf ",h_b[i]);

  }
  
  hipMemcpy(d_a,h_a,bytes,hipMemcpyHostToDevice); 
  hipMemcpy(d_b,h_b,bytes,hipMemcpyHostToDevice); 

  int blockSize=2;

  int gridSize=(int)ceil((float)n/blockSize);

  add<<<gridSize,blockSize>>>(d_a,d_b,d_c,n);

  hipMemcpy(h_c,d_c,bytes,hipMemcpyDeviceToHost);

  printf("\n\nVector BC=>");
  for(i=0;i<n;i++)
  {
   
    printf("%lf ",h_c[i]);

  }

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
   
  free(h_a);
  free(h_b);
  free(h_c);
   
  return 0;
  


}

