#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#include<math.h>

#define N 10



__global__ void sum(double *a,double *o)
{
  int i;
  int id=threadIdx.x;
  for(i=0; i< N ;i++)
  {
     if(id<i)
   {
     a[id]+=a[id+i];
   } 
  }

  o[0]=a[0];
}


__global__ void standardDeviation(double *a,double avg)
{
  
   int id=threadIdx.x;
  
   if(id < N)
   {
      a[id] -= avg;
      a[id]  = a[id] * a[id];

   }
    
}

int main()
{
  
  double *h_a,*d_a,*oh_a,*od_a,*d_a1;
  int size= N * sizeof(double);
  
  h_a=(double *)malloc(size);
  oh_a=(double*)malloc(size);

  hipMalloc(&d_a,size);
  hipMalloc(&d_a1,size);
  hipMalloc(&od_a,size);
  
  int i;
  for(i=0 ;i<N ;i++)
  {
    h_a[i] = random() % N;
  }

  printf("\n\nNumbers =>");
  for(i=0 ;i<N ;i++)
  {
    printf("%lf ",h_a[i]);
  }
  
  hipMemcpy(d_a, h_a,size,hipMemcpyHostToDevice);
  hipMemcpy(d_a1, h_a,size,hipMemcpyHostToDevice);

  sum<<<1, N/2>>>(d_a,od_a);

  hipMemcpy(oh_a, od_a,size,hipMemcpyDeviceToHost);

  printf("\n\nSum => %lf",oh_a[0]);

  float arithmeticMean=(float)oh_a[0]/N;

  printf("\n\nArithmetic Mean => %f",arithmeticMean);

   
  standardDeviation<<<1, N>>>(d_a1,arithmeticMean);

  sum<<<1, N/2>>>(d_a1,od_a);
  
  hipMemcpy(oh_a, od_a,size,hipMemcpyDeviceToHost);

  double temp =oh_a[0]/N;

  
  
  printf("\n\nStandard Deviation => %lf\n\n",sqrt(temp));
  

  hipFree(d_a);
  hipFree(od_a);
  hipFree(d_a1);
  free(h_a);
  free(oh_a);
  
  return 0;  
  
}
