#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>

#define N 9

__global__ void sum(int *a,int *o)
{
  int of;
  int id=threadIdx.x;
  for(of=N/2 ; of > 0 ;of=of/2)
  {
     if(id<of)
   {
     a[id]+=a[id+of];
   } 
  }
  if(N%2==1)
  {
   a[0]=a[0]+a[N-1];
  }
  o[0]=a[0];
}

int main()
{
  
  int *h_a,*d_a,*oh_a,*od_a;
  int size= N * sizeof(int);
  
  h_a=(int*)malloc(size);
  oh_a=(int*)malloc(size);

  hipMalloc(&d_a,size);
  hipMalloc(&od_a,size);

  int i;
  for(i=0 ;i<N ;i++)
  {
    h_a[i] = random() % N;
  }

  printf("\n\nNumbers =>");
  for(i=0 ;i<N ;i++)
  {
    printf("%d ",h_a[i]);
  }
  
  hipMemcpy(d_a, h_a,size,hipMemcpyHostToDevice);

  sum<<<1, N/2>>>(d_a,od_a);

  hipMemcpy(oh_a, od_a,size,hipMemcpyDeviceToHost);

  printf("\n\nSum => %d",oh_a[0]);

  float arithmeticMean=(float)oh_a[0]/N;

  printf("\n\nArithmetic Mean => %f",arithmeticMean);

  hipFree(d_a);
  hipFree(od_a);
  free(h_a);
  free(oh_a); 

  return 0;  
  
}
