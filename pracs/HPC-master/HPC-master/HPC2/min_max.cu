#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h>
#define N 5

__global__ void maximumElement(int *a,int *o)
{

  
    int of;

    int id = threadIdx.x;

     for(of=N/2; of>0;of=of/2)
     {
	if(id<of)
	{
		if(a[id+of] > a[id])
		{
			a[id] = a[id+of];
		}
	}
     }
	if(a[0]<a[N-1])
	{
		a[0]=a[N-1];
	}
        o[0] = a[0];
  
}

__global__ void minimumElement(int *a,int *o)
{

    int of;

    int id = threadIdx.x;

     for(of=N/2;of>0;of=of/2)
     {
	if(id<of)
	{
		if(a[id+of] < a[id])
		{
			a[id] = a[id+of];
		}
	}
     }
	if(a[0]>a[N-1])
	{
		a[0]=a[N-1];
	}
       o[0] = a[0];
  
}




int main()
{
   int *host,*device,*output_host,*output_device;
   int choice;

   int size=N*sizeof(int);
   
   host = (int*)malloc(size);
   output_host = (int*)malloc(size);

   hipMalloc(&device,size);
   hipMalloc(&output_device,size);


   int i;
   /*
   for(i=0 ; i<N ;i++)
   {

     host[i] = random() %N;
 
   }*/

   host[0]=7;
host[1]=2;
host[2]=6;
host[3]=3;
host[4]=1;

   printf("\n\n Vector  => ");
   for(i=0 ; i<N ;i++)
   {

     printf("%d ",host[i]);
     
   }

   hipMemcpy(device,host,size,hipMemcpyHostToDevice);
   
   printf("\n\n1.Maximum Elemnt\n2.Minimum Elemnt\n\nEnter your choice :");
    scanf("%d",&choice);

   if(choice==1)
   {

    maximumElement<<<2,N/2>>>(device,output_device);   

   }
   else
   {
    
    minimumElement<<<2,N/2>>>(device,output_device);

   }

   hipMemcpy(output_host,output_device,size,hipMemcpyDeviceToHost);
     
   if(choice==1)
   {

    printf("\n\nMaximum elemnt => %d",output_host[0]); 
    
   }
   else
   {

   printf("\n\nMinimum elemnt => %d",output_host[0]); 

   }
    
   hipFree(device);
   hipFree(output_device);
   free(host);
   free(output_host);
   
   return 0; 

}
